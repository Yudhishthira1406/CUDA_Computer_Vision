#include "ConvNet.h"

ConvNet::ConvNet(int m, int n, int o) {
    filter_size = m;
    num_of_filters = n;
    output_size = o;


    float init_bias[n];
    float init_weight[n * m];

    for(int i = 0; i < n; ++i) {
        init_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);
        for(int j = 0; j < m; ++j) {
            init_weight[i * m + j] = 0.5f - float(rand()) / float(RAND_MAX);
        }
    }

    hipMalloc(&output, sizeof(float) * output_size);
    hipMalloc(&middle, sizeof(float) * output_size);
    hipMalloc(&bias, sizeof(float) * num_of_filters);
    hipMalloc(&weight, sizeof(float) * filter_size * num_of_filters);
    hipMalloc(&d_output, sizeof(float) * output_size);
    hipMalloc(&d_middle, sizeof(float) * output_size);
    hipMalloc(&d_weight, sizeof(float) * filter_size * num_of_filters);

    hipMemcpy(bias, init_bias, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(weight, init_weight, sizeof(float) * m * n, hipMemcpyHostToDevice);
}


ConvNet::~ConvNet() {
    hipFree(output);
    hipFree(middle);

    hipFree(bias);

    hipFree(weight);

    hipFree(d_output);
    hipFree(d_middle);
    hipFree(d_weight);
}

void ConvNet::reinit() {
    hipMemset(output, 0, sizeof(float) * output_size);
    hipMemset(middle, 0, sizeof(float) * output_size);
}

void ConvNet::reinit_backprop() {
    hipMemset(d_output, 0, sizeof(float) * output_size);
    hipMemset(d_middle, 0, sizeof(float) * output_size);
    hipMemset(d_weight, 0, sizeof(float) * filter_size * num_of_filters);
}